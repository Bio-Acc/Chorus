#include "hip/hip_runtime.h"
#include "cudaTool.h"

#define CUDA_CALL(F)                                                          \
    if ((F) != hipSuccess)                                                   \
    {                                                                         \
        printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
               __FILE__, __LINE__);                                           \
        exit(-1);                                                             \
    }
double getVRAM(int defaultIndexOfGPU)
{
    int deviceCount;
    CUDA_CALL(hipGetDeviceCount(&deviceCount));
    if (defaultIndexOfGPU >= deviceCount) {
        std::cerr << "No CUDA-enabled devices found." << std::endl;
        return 1;
    }
    CUDA_CALL(hipSetDevice(defaultIndexOfGPU));
    size_t freeMem, totalMem;
    CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
    double result = (double)freeMem / (1024 * 1024 * 1024) / 3;
    return max(1.0, result); 
}